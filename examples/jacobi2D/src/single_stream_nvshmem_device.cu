#include "hip/hip_runtime.h"


#include "../include/common.hpp"

typedef float real;

template <int GRID_DIM_X>
__global__ void jacobi_kernel_p2p_full(real *__restrict__ const a_new, const real *__restrict__ const a,
                                       const int iy_start, const int iy_end, const int nx,
                                       real *__restrict__ const a_comm, real *__restrict__ const a_new_comm,
                                       uint64_t *__restrict__ const sync_arr, const int top_pe, const int bottom_pe,
                                       const uint64_t iter) {
    cg::thread_block cta = cg::this_thread_block();
    cg::grid_group grid = cg::this_grid();
    if (blockIdx.x == gridDim.x - 1) {
        for (int ix = threadIdx.y * blockDim.x + threadIdx.x + 1; ix < (nx - 1); ix += blockDim.y * blockDim.x) {
            const real first_row_val = 0.25 * (a[iy_start * nx + ix + 1] + a[iy_start * nx + ix - 1] +
                                               a[(iy_start + 1) * nx + ix] + a_comm[ix]);
            a_new[iy_start * nx + ix] = first_row_val;
            a_comm[ix] = first_row_val;
        }
        nvshmemx_float_put_signal_nbi_block(a_new_comm, a_comm, nx, sync_arr + 1, 1, NVSHMEM_SIGNAL_ADD, top_pe);
        if (threadIdx.y * blockDim.x + threadIdx.x == 0) {
            nvshmem_signal_wait_until(sync_arr, NVSHMEM_CMP_GE, iter + 1);
        }
        cta.sync();
    } else if (blockIdx.x == gridDim.x - 2) {
        for (int ix = threadIdx.y * blockDim.x + threadIdx.x + 1; ix < (nx - 1); ix += blockDim.y * blockDim.x) {
            const real last_row_val = 0.25 * (a[(iy_end - 1) * nx + ix + 1] + a[(iy_end - 1) * nx + ix - 1] +
                                              a_comm[nx + ix] + a[(iy_end - 2) * nx + ix]);
            a_new[(iy_end - 1) * nx + ix] = last_row_val;
            a_comm[nx + ix] = last_row_val;
        }
        nvshmemx_float_put_signal_nbi_block(a_new_comm + nx, a_comm + nx, nx, sync_arr, 1, NVSHMEM_SIGNAL_ADD,
                                            bottom_pe);
        if (threadIdx.y * blockDim.x + threadIdx.x == 0) {
            nvshmem_signal_wait_until(sync_arr + 1, NVSHMEM_CMP_GE, iter + 1);
        }
        cta.sync();
    }
    for (int iy = ((blockIdx.x / GRID_DIM_X) * blockDim.y + threadIdx.y + iy_start + 1); iy < (iy_end - 1);
         iy += (gridDim.x / GRID_DIM_X) * blockDim.y) {
        for (int ix = ((blockIdx.x % GRID_DIM_X) * blockDim.x + threadIdx.x + 1); ix < (nx - 1);
             ix += GRID_DIM_X * blockDim.x) {
            a_new[iy * nx + ix] =
                0.25 * (a[iy * nx + ix + 1] + a[iy * nx + ix - 1] + a[(iy + 1) * nx + ix] + a[(iy - 1) * nx + ix]);
        }
    }
}

int main(int argc, char *argv[]) {
    const uint64_t iter_max = get_argval<uint64_t>(argv, argv + argc, "-niter", DEFAULT_ITER_NUM);
    const uint64_t warmup = get_argval<uint64_t>(argv, argv + argc, "-nwarmup", DEFAULT_SKIP_NUM);
    const int nx = get_argval<int>(argv, argv + argc, "-nx", DEFAULT_NUM_ROW);
    const int ny = get_argval<int>(argv, argv + argc, "-ny", DEFAULT_NUM_COL);

    real *a;
    real *a_new;
    real *a_comm;
    real *a_new_comm;
    uint64_t *sync_arr = nullptr;

    long long unsigned int mesh_size_per_rank = nx * 2;
    long long unsigned int required_symmetric_heap_size = 2 * mesh_size_per_rank * sizeof(real) * 1.1;

    char symmetric_heap_size_str[100];
    sprintf(symmetric_heap_size_str, "%llu", required_symmetric_heap_size);
    setenv("NVSHMEM_SYMMETRIC_SIZE", symmetric_heap_size_str, 1);

    MPI_CALL(MPI_Init(&argc, &argv));
    int rank;
    MPI_CALL(MPI_Comm_rank(MPI_COMM_WORLD, &rank));
    int size;
    MPI_CALL(MPI_Comm_size(MPI_COMM_WORLD, &size));
    int num_devices = 0;
    GPU_RT_CALL(UncGpuGetDeviceCount(&num_devices));

    int local_rank = -1;
    int local_size = 0;
    {
        MPI_Comm local_comm;
        MPI_CALL(MPI_Comm_split_type(MPI_COMM_WORLD, MPI_COMM_TYPE_SHARED, rank, MPI_INFO_NULL, &local_comm));

        MPI_CALL(MPI_Comm_rank(local_comm, &local_rank));
        MPI_CALL(MPI_Comm_size(local_comm, &local_size));

        MPI_CALL(MPI_Comm_free(&local_comm));
    }

    if (1 < num_devices && num_devices < local_size) {
        fprintf(stderr, "ERROR Number of visible devices (%d) is less than number of ranks on the node (%d)!\n",
                num_devices, local_size);
        MPI_CALL(MPI_Finalize());
        return EXIT_FAILURE;
    }

    GPU_RT_CALL(UncGpuSetDevice(local_rank % num_devices));

    nvshmemx_init_attr_t nvshmemInitAttr;
    MPI_Comm mpiInitComm = MPI_COMM_WORLD;
    nvshmemInitAttr.mpi_comm = &mpiInitComm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &nvshmemInitAttr);

    int npes = nvshmem_n_pes();
    int mype = nvshmem_my_pe();

    UncGpuStream_t compute_stream;
    GPU_RT_CALL(UncGpuStreamCreate(&compute_stream));

    UncGpuEvent_t startEvent, stopEvent;
    GPU_RT_CALL(UncGpuEventCreate(&startEvent));
    GPU_RT_CALL(UncGpuEventCreate(&stopEvent));

    int chunk_size;
    int chunk_size_low = (ny - 2) / npes;
    int chunk_size_high = chunk_size_low + 1;

    int num_ranks_low = npes * chunk_size_low + npes - (ny - 2);  // Number of ranks with chunk_size = chunk_size_low
    if (mype < num_ranks_low)
        chunk_size = chunk_size_low;
    else
        chunk_size = chunk_size_high;

    a_comm = static_cast<real *>(nvshmem_malloc(2 * nx * sizeof(real)));
    a_new_comm = static_cast<real *>(nvshmem_malloc(2 * nx * sizeof(real)));
    sync_arr = static_cast<uint64_t *>(nvshmem_malloc(4 * sizeof(uint64_t)));

    GPU_RT_CALL(UncGpuMalloc(&a, nx * chunk_size_high * sizeof(real)));
    GPU_RT_CALL(UncGpuMalloc(&a_new, nx * chunk_size_high * sizeof(real)));

    GPU_RT_CALL(UncGpuMemset(a_comm, 0, nx * 2 * sizeof(real)));
    GPU_RT_CALL(UncGpuMemset(a_new_comm, 0, nx * 2 * sizeof(real)));
    GPU_RT_CALL(UncGpuMemset(sync_arr, 0, 4 * sizeof(uint64_t)));
    GPU_RT_CALL(UncGpuMemset(a, 0, nx * chunk_size * sizeof(real)));
    GPU_RT_CALL(UncGpuMemset(a_new, 0, nx * chunk_size * sizeof(real)));

    // Calculate local domain boundaries
    int iy_start_global;  // My start index in the global array
    if (mype < num_ranks_low) {
        iy_start_global = mype * chunk_size_low;
    } else {
        iy_start_global = num_ranks_low * chunk_size_low + (mype - num_ranks_low) * chunk_size_high;
    }
    int iy_end_global = iy_start_global + chunk_size - 1;  // My last index in the global array
    // do not process boundaries
    iy_end_global = std::min(iy_end_global, ny - 1);

    int iy_start = 0;
    int iy_end = (iy_end_global - iy_start_global + 1) + iy_start;

    int device;
    GPU_RT_CALL(UncGpuGetDevice(&device));
    UncGpuDeviceProp_t deviceProp{};
    GPU_RT_CALL(UncGpuGetDeviceProperties(&deviceProp, device));
    int numSms = deviceProp.multiProcessorCount;

    constexpr int grid_dim_x = 11;
    // const int grid_dim_y = (numSms - 2) / grid_dim_x;

    // calculate boundary indices for top and bottom boundaries
    int top_pe = mype > 0 ? mype - 1 : (npes - 1);
    int bottom_pe = (mype + 1) % npes;

    constexpr int dim_block_x = 32;
    constexpr int dim_block_y = 32;
    uint64_t iter = 0;
    void *kernelArgs[] = {(void *)&a_new,  (void *)&a,         (void *)&iy_start,   (void *)&iy_end,
                          (void *)&nx,     (void *)&a_comm,    (void *)&a_new_comm, (void *)&sync_arr,
                          (void *)&top_pe, (void *)&bottom_pe, (void *)&iter};

    // Set diriclet boundary conditions on left and right boundary
    initialize_boundaries<<<(ny / npes) / 128 + 1, 128, 0, compute_stream>>>(a, a_new, PI, iy_start_global, nx, iy_end,
                                                                             ny);
    GPU_RT_CALL(UncGpuMemsetAsync(sync_arr, 0, 4 * sizeof(uint64_t), compute_stream));
    // comm.Barrier(compute_stream);
    GPU_RT_CALL(UncGpuDeviceSynchronize());
    MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));

    PUSH_RANGE("Jacobi solve", 0)
    for (iter = 0; iter < iter_max + warmup; ++iter) {
        if (mype == 0 && iter == warmup) {
            GPU_RT_CALL(UncGpuEventRecord(startEvent, compute_stream));
        }
        // jacobi_step.LaunchKernel();
        nvshmemx_collective_launch((void *)jacobi_kernel_p2p_full<grid_dim_x>, numSms,
                                   dim3(dim_block_x, dim_block_y, 1), kernelArgs, 0, compute_stream);

        std::swap(a_new, a);
        std::swap(a_new_comm, a_comm);
    }
    // comm.Barrier(compute_stream);
    nvshmemx_barrier_all_on_stream(compute_stream);
    if (mype == 0) {
        GPU_RT_CALL(UncGpuEventRecord(stopEvent, compute_stream));
        GPU_RT_CALL(UncGpuEventSynchronize(stopEvent));
        POP_RANGE
        float multi_gpu_time = 0.0;
        GPU_RT_CALL(UncGpuEventElapsedTime(&multi_gpu_time, startEvent, stopEvent));
        printf("jacobi, nvshmem_d, %d, %f\n", npes, multi_gpu_time);
    }

    GPU_RT_CALL(UncGpuDeviceSynchronize());
    MPI_CALL(MPI_Barrier(MPI_COMM_WORLD));
    GPU_RT_CALL(UncGpuFree(a));
    GPU_RT_CALL(UncGpuFree(a_new));
    nvshmem_free(a_comm);
    nvshmem_free(a_new_comm);
    nvshmem_free(sync_arr);
    GPU_RT_CALL(UncGpuEventDestroy(startEvent));
    GPU_RT_CALL(UncGpuEventDestroy(stopEvent));
    GPU_RT_CALL(UncGpuStreamDestroy(compute_stream));

    nvshmem_finalize();
    MPI_CALL(MPI_Finalize());
    return 0;
}
